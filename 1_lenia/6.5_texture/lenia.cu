
#include <hip/hip_runtime.h>
#include <ctype.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <math.h>
#include <time.h>
#include <string.h>

extern char *optarg;
extern int optopt;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__host__ __device__ float bell(const float val, const float mean, const float std)
{
    const float t = (val - mean) / std;
    return expf(-(t * t) / 2.f);
}

__host__ __device__ float clamp(float d, float min, float max) {
  const float t = d < min ? min : d;
  return t > max ? max : t;
}

void init_kernel(float *const kernel, const size_t size, const size_t R, const float mean, const float std)
{
    double sum = 0;
    for (size_t i = 0; i < size; ++i) {
        for (size_t j = 0; j < size; ++j) {
            double res = 0;
            double dist2 = hypot((double)i - R, (double)j - R);
            if (dist2 < R) {
                res = bell(dist2 / R, mean, std);
            }
            sum += res;
            kernel[i * size + j] = res;
        }
    }    
    for (size_t i = 0; i < size; ++i) {
        for (size_t j = 0; j < size; ++j) {
            kernel[i * size + j] /= sum;
        }
    }
}

__host__ void pad_field(
    float const *const __restrict__ src, float *const __restrict__ dst, const size_t src_size, const size_t pad_size, hipStream_t streams[3]
)
{
    const size_t PADDED_SIZE = src_size + 2 * pad_size;
    gpuErrchk(hipMemcpy2DAsync(&dst[pad_size * PADDED_SIZE + pad_size], PADDED_SIZE * sizeof(float), src, src_size * sizeof(float), src_size * sizeof(float), src_size, hipMemcpyDeviceToDevice, streams[0]));
    gpuErrchk(hipMemcpy2DAsync(&dst[pad_size * PADDED_SIZE], PADDED_SIZE * sizeof(float), &src[src_size - pad_size], src_size * sizeof(float), pad_size * sizeof(float), src_size, hipMemcpyDeviceToDevice, streams[1]));
    gpuErrchk(hipMemcpy2DAsync(&dst[pad_size * PADDED_SIZE + pad_size + src_size], PADDED_SIZE * sizeof(float), src, src_size * sizeof(float), pad_size * sizeof(float), src_size, hipMemcpyDeviceToDevice, streams[2]));
    gpuErrchk(hipStreamSynchronize(streams[0]));
    gpuErrchk(hipStreamSynchronize(streams[1]));
    gpuErrchk(hipStreamSynchronize(streams[2]));
    gpuErrchk(hipMemcpyAsync(&dst[0], &dst[src_size * PADDED_SIZE], PADDED_SIZE * pad_size * sizeof(float), hipMemcpyDeviceToDevice, streams[0]));
    gpuErrchk(hipMemcpyAsync(&dst[(pad_size + src_size)* PADDED_SIZE], &dst[pad_size * PADDED_SIZE], PADDED_SIZE * pad_size * sizeof(float), hipMemcpyDeviceToDevice, streams[1]));
    gpuErrchk(hipStreamSynchronize(streams[0]));
    gpuErrchk(hipStreamSynchronize(streams[1]));
}

__global__ void make_step(
        hipTextureObject_t current_state,
        float *const __restrict__ next_state, const size_t size,
        float const *const __restrict__ kernel, const size_t kernel_size,
        const float dT, const float growth_mean, const float growth_std
)
{
    extern __shared__ char shared_data[];
    float * const __restrict__ kernel_shared = (float*)shared_data;

    const size_t thread_pos_x = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t thread_pos_y = blockIdx.y * blockDim.y + threadIdx.y;
    if (thread_pos_x >= size || thread_pos_y >= size) {
        return;
    }
    const size_t BLOCK_SIZE_FLAT = blockDim.x * blockDim.y;
    const size_t KERNEL_DATA_SIZE = kernel_size * kernel_size;
    const size_t KERNEL_NCOPY = (KERNEL_DATA_SIZE + BLOCK_SIZE_FLAT - 1) / BLOCK_SIZE_FLAT;
    const size_t THREAD_IDX = threadIdx.x + blockDim.x * threadIdx.y;
    for (size_t i = 0, p = THREAD_IDX * KERNEL_NCOPY; i < KERNEL_NCOPY && p < KERNEL_DATA_SIZE; ++i, ++p) {
        kernel_shared[p] = kernel[p];
    }

    __syncthreads();

    float next_state_val = 0;
    const size_t R = (kernel_size - 1) / 2;
    for (size_t ki = 0; ki < kernel_size; ++ki) {
        const size_t ki_offset = ki * kernel_size;
        for (size_t kj = 0; kj < kernel_size; ++kj) {
            next_state_val += kernel_shared[ki_offset + kj] * tex2D<float>(current_state, thread_pos_x + kj + size - R, thread_pos_y + ki + size - R);
        }
    }

    next_state_val = clamp(tex2D<float>(current_state, thread_pos_x, thread_pos_y) + 0 * dT * (bell(next_state_val, growth_mean, growth_std) * 2 - 1), 0, 1);
    next_state[thread_pos_y * size + thread_pos_x] = next_state_val;
}

int main(int argc, char *argv[])
{
    size_t nsteps = 32;
    char const *input_path = "input.bin";
    char const *output_path = "output.bin";

    int c;
    while ((c = getopt(argc, argv, "i:s:o:")) != -1)
    {
        switch (c)
        {
            char *endptr;
            case 's':
                nsteps = strtoul(optarg, &endptr, 10);
                break;
            case 'i':
                input_path = optarg;
                break;
            case 'o':
                output_path = optarg;
                break;
            case '?':
                printf("Unknown option `-%c'.\n", optopt);
                return 1;
            default:
                abort();
        }
    }

    FILE *inFile = fopen(input_path, "rb");
    
    size_t FIELD_SIZE = 256;
    float dT = 0.1;
    size_t R = 13;  // cells per kernel radius
    float GROWSH_MEAN = 0.15;
    float GROWSH_STD = 0.015;

    size_t FIELD_DATA_SIZE, RESULT_DATA_SIZE;
    float *result = NULL;
    if (inFile != NULL) {
        if (!(
            fread(&FIELD_SIZE, sizeof(size_t), 1, inFile)
            && fread(&dT, sizeof(float), 1, inFile)
            && fread(&R, sizeof(size_t), 1, inFile)
            && fread(&GROWSH_MEAN, sizeof(float), 1, inFile)
            && fread(&GROWSH_STD, sizeof(float), 1, inFile)
        )) {
            printf("Cannot read parameters\n");
            abort();
        }

        FIELD_DATA_SIZE = FIELD_SIZE * FIELD_SIZE;
        RESULT_DATA_SIZE = FIELD_DATA_SIZE * (nsteps + 1);
        hipHostAlloc(&result, RESULT_DATA_SIZE * sizeof(float), 0);

        if (!fread(result, FIELD_DATA_SIZE * sizeof(float), 1, inFile)) {
            printf("Cannot read field\n");
            abort();
        }
        fclose(inFile);
    }
    else {
        printf("Cannot open file %s\n" , input_path);
        abort();
    }

    const float KERNEL_MEAN = 0.5;
    const float KERNEL_STD = 0.15;
    const size_t KERNEL_SIZE = 2 * R + 1;

    float *const kernel = (float*)calloc(KERNEL_SIZE * KERNEL_SIZE, sizeof(float));
    init_kernel(kernel, KERNEL_SIZE, R, KERNEL_MEAN, KERNEL_STD);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat); 

    hipArray *texArray;
    hipMallocArray(&texArray, &channelDesc, FIELD_SIZE, FIELD_SIZE);

    float *kernel_gpu;
    gpuErrchk(hipMalloc(&kernel_gpu, KERNEL_SIZE * KERNEL_SIZE * sizeof(float)));
    gpuErrchk(hipMemcpy(kernel_gpu, kernel, KERNEL_SIZE * KERNEL_SIZE * sizeof(float), hipMemcpyHostToDevice));

    float *current_field_gpu, *next_field_gpu, *current_field_padded_gpu;
    gpuErrchk(hipMalloc(&current_field_gpu, FIELD_DATA_SIZE * sizeof(float)));
    gpuErrchk(hipMemcpy(current_field_gpu, result, FIELD_DATA_SIZE * sizeof(float), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&next_field_gpu, FIELD_DATA_SIZE * sizeof(float)));
    gpuErrchk(hipMemset(next_field_gpu, 0, FIELD_DATA_SIZE * sizeof(float)));

    const size_t PADDED_SIZE = FIELD_SIZE + 2 * R;
    gpuErrchk(hipMalloc(&current_field_padded_gpu, PADDED_SIZE * PADDED_SIZE * sizeof(float)));

    dim3 blocks_shape = dim3(24, 32);
    dim3 grid_shape  = dim3((FIELD_SIZE + blocks_shape.x - 1) / blocks_shape.x, (FIELD_SIZE + blocks_shape.y - 1) / blocks_shape.y);

    hipStream_t pad_streams[3], kernel_stream, copy_stream;
    gpuErrchk(hipStreamCreate(&pad_streams[0]));
    gpuErrchk(hipStreamCreate(&pad_streams[1]));
    gpuErrchk(hipStreamCreate(&pad_streams[2]));
    gpuErrchk(hipStreamCreate(&kernel_stream));
    gpuErrchk(hipStreamCreate(&copy_stream));

    for (size_t it = 0; it < nsteps; ++it) {
        gpuErrchk(hipMemcpy2DToArrayAsync(texArray, 0, 0, current_field_gpu, FIELD_SIZE * sizeof(float), FIELD_SIZE * sizeof(float), FIELD_SIZE, hipMemcpyDeviceToDevice, kernel_stream));
        
        struct hipResourceDesc resDesc;
        memset(&resDesc, 0, sizeof(resDesc));
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = texArray;

        struct hipTextureDesc texDesc;
        memset(&texDesc, 0, sizeof(texDesc));
        texDesc.addressMode[0] = hipAddressModeWrap;
        texDesc.addressMode[1] = hipAddressModeWrap;
        texDesc.filterMode = hipFilterModePoint;
        texDesc.readMode = hipReadModeElementType;
        texDesc.normalizedCoords = 0;

        hipTextureObject_t texObj = 0;
        gpuErrchk(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL));


        float *const next_state = &result[(it+1) * FIELD_DATA_SIZE];
        make_step<<<grid_shape, blocks_shape, KERNEL_SIZE*KERNEL_SIZE*sizeof(int), kernel_stream>>>(texObj, next_field_gpu, FIELD_SIZE, kernel_gpu, KERNEL_SIZE, dT, GROWSH_MEAN, GROWSH_STD);
        gpuErrchk(hipStreamSynchronize(kernel_stream));
        gpuErrchk(hipStreamSynchronize(copy_stream));
        gpuErrchk(hipMemcpyAsync(next_state, next_field_gpu, FIELD_DATA_SIZE * sizeof(float), hipMemcpyDeviceToHost, copy_stream));
        float *tmp = current_field_gpu;
        current_field_gpu = next_field_gpu;
        next_field_gpu = tmp;
        gpuErrchk(hipMemsetAsync(next_field_gpu, 0, FIELD_DATA_SIZE * sizeof(float), kernel_stream));
        gpuErrchk(hipDestroyTextureObject(texObj));
    }
    gpuErrchk(hipStreamSynchronize(copy_stream));
    hipEventRecord(stop);
    hipDeviceSynchronize();
    float time_diff_ms = 0;
    gpuErrchk(hipEventElapsedTime(&time_diff_ms, start, stop));
    fprintf(stderr, "Program took %.4f ms\n%.8f ms per iteration\n", time_diff_ms, time_diff_ms / nsteps);
    FILE *outFile = fopen(output_path, "wb");
    if (outFile != NULL) {
        fwrite(&FIELD_SIZE, sizeof(size_t), 1, outFile);
        fwrite(&nsteps, sizeof(size_t), 1, outFile);
        fwrite(result, RESULT_DATA_SIZE * sizeof(float), 1, outFile);
        fclose(outFile);
    }
    hipFree(kernel_gpu);
    hipFree(current_field_gpu);
    hipFree(next_field_gpu);
    hipFree(current_field_padded_gpu);
    free(kernel);
    hipHostFree(result);
}